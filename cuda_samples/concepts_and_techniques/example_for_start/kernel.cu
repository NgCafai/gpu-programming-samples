#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "kernel.h"

__global__ void addKernel(int *c, const int *a, const int *b, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}

void runAddKernel(int *c, const int *a, const int *b, int size) {
    int *dev_a, *dev_b, *dev_c;

    // Allocate GPU buffers
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));
    hipMalloc((void**)&dev_c, size * sizeof(int));

    // Copy input arrays to the GPU
    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    // Launch the kernel
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b, size);

    // Copy the results back to the host
    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

    // Free the GPU memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}
