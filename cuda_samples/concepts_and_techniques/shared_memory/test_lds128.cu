
#include <hip/hip_runtime.h>
#include <cstdint>
#include <cstdio>

__global__ void smem_1(uint32_t *a) {
  __shared__ uint32_t smem[128];
  uint32_t tid = threadIdx.x;
  for (int i = 0; i < 4; i++) {
    smem[i * 32 + tid] = tid;
  }
  __syncthreads();
  if (tid == 15 || tid == 16) {
    reinterpret_cast<uint4 *>(a)[tid] =
        reinterpret_cast<const uint4 *>(smem)[4];
  }
}

__global__ void smem_2(uint32_t *a) {
  __shared__ uint32_t smem[128];
  uint32_t tid = threadIdx.x;
  for (int i = 0; i < 4; i++) {
    smem[i * 32 + tid] = tid;
  }
  __syncthreads();
  if (tid == 0 || tid == 15) {
    reinterpret_cast<uint4 *>(a)[tid] =
        reinterpret_cast<const uint4 *>(smem)[4];
  }
}

__global__ void smem_3(uint32_t *a) {
  __shared__ uint32_t smem[128];
  uint32_t tid = threadIdx.x;
  for (int i = 0; i < 4; i++) {
    smem[i * 32 + tid] = tid;
  }
  __syncthreads();
  reinterpret_cast<uint4 *>(a)[tid] = reinterpret_cast<const uint4 *>(
      smem)[(tid / 8) * 2 + ((tid % 8) / 2) % 2];
}

__global__ void smem_4(uint32_t *a) {
  __shared__ uint32_t smem[128];
  uint32_t tid = threadIdx.x;
  for (int i = 0; i < 4; i++) {
    smem[i * 32 + tid] = tid;
  }
  __syncthreads();
  uint32_t addr;
  if (tid < 16) {
    addr = (tid / 8) * 2 + ((tid % 8) / 2) % 2;
  } else {
    addr = (tid / 8) * 2 + ((tid % 8) % 2);
  }
  reinterpret_cast<uint4 *>(a)[tid] =
      reinterpret_cast<const uint4 *>(smem)[addr];
  // printf("tid: %d, addr: %d\n", tid, addr);
}

__global__ void smem_5(uint32_t *a) {
  __shared__ uint32_t smem[128];
  uint32_t tid = threadIdx.x;
  for (int i = 0; i < 4; i++) {
    smem[i * 32 + tid] = tid;
  }
  __syncthreads();
  uint32_t addr = (tid / 16) * 4 + (tid % 16) / 8 + (tid % 8) / 4 * 8;
  reinterpret_cast<uint4 *>(a)[tid] =
      reinterpret_cast<const uint4 *>(smem)[addr];
  printf("tid: %d, addr: %d\n", tid, addr);
}


int main() {
  uint32_t *d_a;
  hipMalloc(&d_a, sizeof(uint32_t) * 128);
  // smem_1<<<1, 32>>>(d_a);
  // smem_2<<<1, 32>>>(d_a);
  // smem_3<<<1, 32>>>(d_a);
  // smem_4<<<1, 32>>>(d_a);
  smem_5<<<1, 32>>>(d_a);
  hipFree(d_a);
  hipDeviceSynchronize();
  return 0;
}