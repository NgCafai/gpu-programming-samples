#include <cstdio>
#include <cstdlib>
#include <string>
#include <vector>

#include "src/runner.cuh"
#include "src/utils.cuh"

const int ALPHA = 1;
const int BETA = 0;
const int REPEAT_TIMES = 50;
const std::vector<int> SIZE = {128, 256, 512, 1024, 2048, 4096};

int main(int argc, char **argv) {
    if (argc != 2) {
        printf("Please select a kernel version (range 0 - 3, 0 for NVIDIA cuBLAS)\n");
        exit(EXIT_FAILURE);
    }

    // get kernel_version
    int kernel_version = std::stoi(argv[1]);
    if (kernel_version < 0 || kernel_version > 3) {
        printf(
            "Please enter a valid kernel version (range 0 - 3, 0 for NVIDIA cuBLAS)\n");
        exit(EXIT_FAILURE);
    }

    // get devide_idx
    int devide_idx = 0;
    if (argc == 3) {
        devide_idx = std::stoi(argv[2]);
    }

    int M, N, K;
    float *C = nullptr, *C_cublas = nullptr;  // host matrices
    float *d_A = nullptr, *d_B = nullptr, *d_C = nullptr,
          *d_C_cublas = nullptr;  // device matrices

    

    float elapsed_time;
    

    // run the kernel
    printf("Running kernel version %d on device %d\n", kernel_version, devide_idx);

    for (auto size : SIZE) {
        cudaErrCheck(hipSetDevice(devide_idx));

        hipblasHandle_t blas_handle;
        cublasErrCheck(hipblasCreate(&blas_handle));

        hipEvent_t start, end;
        cudaErrCheck(hipEventCreate(&start));
        cudaErrCheck(hipEventCreate(&end));

        M = N = K = size;

        // allocate host memory
        C = (float *)malloc(M * N * sizeof(float));
        C_cublas = (float *)malloc(M * N * sizeof(float));

        // allocate device memory
        cudaErrCheck(hipMalloc((void **)&d_A, M * K * sizeof(float)));
        cudaErrCheck(hipMalloc((void **)&d_B, K * N * sizeof(float)));
        cudaErrCheck(hipMalloc((void **)&d_C, M * N * sizeof(float)));
        cudaErrCheck(hipMalloc((void **)&d_C_cublas, M * N * sizeof(float)));

        // generate data
        hiprandGenerator_t generator;
        curandErrCheck(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT));
        curandErrCheck(hiprandSetPseudoRandomGeneratorSeed(generator, 1337ULL));

        curandErrCheck(hiprandGenerateUniform(generator, d_A, M * K));
        curandErrCheck(hiprandGenerateUniform(generator, d_B, K * N));

        curandErrCheck(hiprandDestroyGenerator(generator));

        // ----------------------------------------------------------------------------
        // verify the correctness of the kernel && warm up so as to avoid the first-time
        // overhead overhead
        // 1) self-implemented kernel
        RunSgemmKernel(kernel_version, M, N, K, ALPHA, d_A, d_B, BETA, d_C, blas_handle);
        cudaErrCheck(hipDeviceSynchronize());
        // 2) cuBLAS
        RunSgemmKernel(0, M, N, K, ALPHA, d_A, d_B, BETA, d_C_cublas, blas_handle);
        cudaErrCheck(hipDeviceSynchronize());
        hipMemcpy(C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(C_cublas, d_C_cublas, M * N * sizeof(float), hipMemcpyDeviceToHost);
        if (!IsMatrixEqual(C, C_cublas, M, N)) {
            printf("Matrix mismatch\n");

            if (size <= 128) {
                std::string err_file = "./debug/err_" + std::to_string(size) + ".txt";
                printf("Writing matrix to %s\n", err_file.c_str());
                std::ofstream out(err_file);
                out << "Matrix C: \n";
                PrintMatrix(C, M, N, out);
                out << "Matrix C_cublas: \n";
                PrintMatrix(C_cublas, M, N, out);
            }
            exit(EXIT_FAILURE);
        }

        // ----------------------------------------------------------------------------
        // measure the performance
        cudaErrCheck(hipEventRecord(start));
        for (int i = 0; i < REPEAT_TIMES; i++) {
            RunSgemmKernel(kernel_version, M, N, K, ALPHA, d_A, d_B, BETA, d_C, blas_handle);
        }
        cudaErrCheck(hipEventRecord(end));
        cudaErrCheck(hipEventSynchronize(end));
        cudaErrCheck(hipEventElapsedTime(&elapsed_time, start, end));
        elapsed_time /= 1000.0f;  // convert ms to s

        long long total_flop = static_cast<long long>(M) * N * K * REPEAT_TIMES * 2;
        double avg_elapsed_time = double(elapsed_time) / REPEAT_TIMES;
        double gflops = (double)total_flop / 1e9 / double(elapsed_time);
        printf("Size: %d, Average elapsed time: %7.6f s, Performance: %7.2f GFLOPS. \n",
               size, avg_elapsed_time, gflops);
        fflush(stdout);

        // free memory
        free(C);
        free(C_cublas);

        cudaErrCheck(hipFree(d_A));
        cudaErrCheck(hipFree(d_B));
        cudaErrCheck(hipFree(d_C));
        cudaErrCheck(hipFree(d_C_cublas));

        cublasErrCheck(hipblasDestroy(blas_handle));
        cudaErrCheck(hipEventDestroy(start));
        cudaErrCheck(hipEventDestroy(end));

        cudaErrCheck(hipDeviceReset());
    }

    return 0;
}